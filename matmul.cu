#include "hip/hip_runtime.h"
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <sys/time.h>

#include <fstream>
#include <iostream>
#include <string>

#include "apis_cu.h"
#include "hip/hip_runtime.h"
#include ""

/**
 * 本示例程序为：通过4个GPU chiplet
 * 计算随机数矩阵A（400 * 100）与随机数矩阵B（100 * 400）相乘结果。
 * 由矩阵乘法原理可知，我们可将计算任务划分为4个100*100的矩阵相乘，并将结果相加。
 */

// #define Row 100
// #define Col 100
int Row_A = 20;
int Col_A = 100;
int Row_B = 100;
int Col_B = 100;

/**
 * 矩阵乘法的核心函数，由每个线程都会运行一次本函数，
 * 根据线程编号不同计算出位于结果矩阵不同位置的数据。
 */

__global__ void matrix_mul_gpu(int64_t *M, int64_t *N, int64_t *P, int64_t widthA, int64_t heightA,
                               int64_t widthB) {
    int64_t i = threadIdx.x + blockDim.x * blockIdx.x;
    int64_t j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < widthB && j < heightA) {
        int64_t sum = 0;
        for (int64_t k = 0; k < widthA; k++) {
            int64_t a = M[j * widthA + k];
            int64_t b = N[k * widthB + i];
            sum += a * b;
        }
        P[j * widthB + i] = sum;
    }
}

int main(int argc, char** argv) {
    // 读取本进程所代表的chiplet编号

    int idX = atoi(argv[1]);
    int idY = atoi(argv[2]);
    int64_t *d_dataA, *d_dataB, *d_dataC;
    hipMalloc((void**)&d_dataA, sizeof(int64_t) * Row_A * Col_A);
    hipMalloc((void**)&d_dataB, sizeof(int64_t) * Row_B * Col_B);
    hipMalloc((void**)&d_dataC, sizeof(int64_t) * Col_B * Row_A);

    receiveMessage(idX, idY, 3, 3, d_dataA, sizeof(int64_t) * Row_A * Col_A);
    receiveMessage(idX, idY, 3, 3, d_dataB, sizeof(int64_t) * Row_B * Col_B);

    // calculate
    dim3 threadPerBlock(10, 10);
    dim3 blockNumber((Col_B + threadPerBlock.x - 1) / threadPerBlock.x,
                         (Row_A + threadPerBlock.y - 1) / threadPerBlock.y);
    matrix_mul_gpu<<<blockNumber, threadPerBlock>>>(d_dataA, d_dataB, d_dataC, Col_A, Row_A, Col_B);

    sendMessage(3, 3, idX, idY, d_dataC, Col_B * Row_A * sizeof(int64_t));
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    return 0;
}
